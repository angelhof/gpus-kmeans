#include <hip/hip_runtime.h>
#include "gpu_util.h"

void *gpu_alloc(size_t count)
{
    void *ret;
    if (hipMalloc(&ret, count) != hipSuccess) {
        ret = NULL;
    }

    return ret;
}

void gpu_free(void *gpuptr)
{
    hipFree(gpuptr);
}

int copy_to_gpu(const void *host, void *gpu, size_t count)
{
    if (hipMemcpy(gpu, host, count, hipMemcpyHostToDevice) != hipSuccess)
        return -1;
    return 0;
}

int copy_from_gpu(void *host, const void *gpu, size_t count)
{
    if (hipMemcpy(host, gpu, count, hipMemcpyDeviceToHost) != hipSuccess)
        return -1;
    return 0;
}

const char *gpu_get_errmsg(hipError_t err)
{
    return hipGetErrorString(err);
}

const char *gpu_get_last_errmsg()
{
    return gpu_get_errmsg(hipGetLastError());
}
    
