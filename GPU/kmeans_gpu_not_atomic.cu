#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "gpu_util.h"
#include "kmeans_util.h"
#include "hipblas.h"

/* gpu parameters */
//#define GRID_SIZE 16
//#define BLOCK_SIZE 256

#if __CUDA_ARCH__ < 600
__device__ double doubleAtomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

int main(int argc, char *argv[]) {
    
    int n, k, i, j;
    int dim = 2;
    double **points;
    
    int BLOCK_SIZE = 256; //Default
    if (argc > 1) BLOCK_SIZE = atoi(argv[1]);
    
    //The second input argument should be the dataset filename
    FILE *in;
    if (argc > 2) {
        in = fopen(argv[2], "r");
    } else {
        in = stdin;
    }

    //Parse file
    register short read_items = -1;
    read_items = fscanf(in, "%d %d %d\n", &n ,&k, &dim);
    if (read_items != 3){
        printf("Something went wrong with reading the parameters!\n");
        return EXIT_FAILURE;
    }
    points = create_2D_double_array(n, dim);
    for (i =0; i<n; i++) {
        for (j=0; j<dim; j++) {
            read_items = fscanf(in, "%lf", &points[i][j]);
            if (read_items != 1) {
                printf("Something went wrong with reading the points!\n");
            }
        }
    }
    fclose(in);
        
    printf("Input Read successfully \n");
    
    //Create CUBLAS Handles
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed!\n");
        return EXIT_FAILURE;
    }
    
    // Calculate grid and block sizes
    int grid_size = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
    dim3 gpu_grid(grid_size, 1);
    dim3 gpu_block(BLOCK_SIZE, 1);
    
    printf("Grid size : %dx%d\n", gpu_grid.x, gpu_grid.y);
    printf("Block size: %dx%d\n", gpu_block.x, gpu_block.y);
    
    clock_t start = clock();
    
    double **centers;
    printf("Initializing Centers...\n");
    centers = init_centers_kpp(points, n, k, dim);
    printf("Initializing Centers done\n");
    
    // start algorithm
    
    // GPU allocations
    double *dev_centers, *dev_points;
    double *dev_new_centers;
    double *dev_points_clusters;
    double *dev_points_in_cluster;
    double *dev_ones;

    dev_centers = (double *) gpu_alloc(k*dim*sizeof(double));
    dev_points = (double *) gpu_alloc(n*dim*sizeof(double));
    dev_points_in_cluster = (double *) gpu_alloc(k*sizeof(double));
    dev_points_clusters = (double *) gpu_alloc(n*k*sizeof(double));
    dev_new_centers = (double *) gpu_alloc(k*dim*sizeof(double));
    dev_ones = (double *) gpu_alloc(n*sizeof(double));
    
    printf("GPU allocs done \n");
    
    call_create_dev_ones(dev_ones, n, gpu_grid, gpu_block);

    // Transpose points and centers for cublas
    // TODO: Transpose at cublas in gpu
    double * staging_points = (double*) calloc(n*dim, sizeof(double));
    double * staging_centers = (double*) calloc(k*dim, sizeof(double));
    transpose(points, staging_points, n, dim);
    transpose(centers, staging_centers, k, dim);

    // Copy points to GPU
    if (copy_to_gpu(staging_points, dev_points, n*dim*sizeof(double)) != 0) {
        printf("Error in copy_to_gpu points\n");
        return -1;
    }

    // Copy centers to GPU
    if (copy_to_gpu(staging_centers, dev_centers, k*dim*sizeof(double)) != 0) {
        printf("Error in copy_to_gpu centers\n");
        return -1;
    }

    int step = 0;
    int check = 0;
    int* dev_check = (int *) gpu_alloc(sizeof(int));

    // Debug
    printf("Initial centers:\n");
    for(i=0;i<k;i++){
        for(j=0;j<dim;j++)
            printf("%lf,\t", centers[i][j]);
        printf("\n");
    }

    printf("Loop Start...\n");
    while (!check) {
        kmeans_on_gpu(
                    dev_points,
                    dev_centers,
                    n, k, dim,
                    dev_points_clusters,
                    dev_points_in_cluster,
                    dev_new_centers,
                    dev_check,
                    BLOCK_SIZE,
                    handle,
                    dev_ones);
        
        copy_from_gpu(&check, dev_check, sizeof(int));
        
        // printf("Step %d Check: %d \n", step, check);
        //if (check < EPS) break;
        
        step += 1;
        // if (step == 3) break;
    }

    printf("Total num. of steps is %d.\n", step);

    double time_elapsed = (double)(clock() - start) / CLOCKS_PER_SEC;
    printf("Total Time Elapsed: %lf seconds\n", time_elapsed);
    
    FILE *f;
    //Store Performance metrics
    //For now just the time elapsed, in the future maybe we'll need memory GPU memory bandwidth etc...
    f = fopen("log.out", "w");
    fprintf(f, "Time Elapsed: %lf ", time_elapsed);
    fclose(f);
    
        
    // print & save results
    
    f = fopen("centers.out", "w");
    
    copy_from_gpu(staging_centers, dev_centers, k*dim*sizeof(double));
    printf("Centers:\n");
    for (i = 0; i < k; i++) {
        for (j = 0; j < dim; j++){
            printf("%lf,\t", staging_centers[j*k + i]);
            fprintf(f, "%lf ", staging_centers[j*k + i]);
        }
        printf("\n");
        fprintf(f, "\n");
    }
    fclose(f);
    
    //Store Mapping Data in case we need it
    // int *points_clusters;
    // points_clusters = (int *)calloc(n, sizeof(int));
    // copy_from_gpu(points_clusters, dev_points_clusters, n*sizeof(int));
    // f = fopen("point_cluster_map.out", "w");
    // for (i =0;i<n;i++){
    //     fprintf(f, "%d\n", points_clusters[i]);
    // }
    
    // fclose(f);
    
    // GPU clean
    gpu_free(dev_centers);
    gpu_free(dev_points);
    gpu_free(dev_points_in_cluster);
    gpu_free(dev_points_clusters);
    gpu_free(dev_ones);

    stat = hipblasDestroy(handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed!\n");
        return EXIT_FAILURE;
    }

    // clear and exit
    delete_points(points);
    delete_points(centers);
    // free(points_clusters);
    return 0;
}
