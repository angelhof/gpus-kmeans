#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu_util.h"
#include "hipblas.h"
#include "kmeans_util.h"

#define MIN(X, Y) (((X) < (Y)) ? (X) : (Y))
#define MAX(X, Y) (((X) > (Y)) ? (X) : (Y))

#ifndef EPS
#   define EPS 1.e-6
#endif

#ifdef DEBUG
#define DPRINTF(fmt, args...) \
do { \
    printf("%s, line %u: " fmt "\r\n", __FUNCTION__, __LINE__ , ##args); \
    fflush(stdout); \
} while (0)
#else   
#define DPRINTF(fmt, args...) do{}while(0)
#endif


__constant__ double dev_centers[MAX_CONSTANT_MEMORY];


__device__ int get_global_tid() {
    return (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x*blockDim.y +
        blockDim.x*threadIdx.y + threadIdx.x;
}

double squared_distance(double* ps, double* center, int dim) {
    double sum = 0;

    for (int i = 0; i < dim; i++){
        double temp = center[i] - ps[i];
        sum += temp * temp;
    }

    return sum;
}

__device__
double squared_distance_on_gpu(const double* ps, const double* center, const int block_size, const int k, const int dim) {
    double sum = 0;

    for (int i = 0, j=0; i < dim*block_size; i+=block_size,j+=k){
        double temp = center[j] - ps[i];
        // sum = temp * temp + sum as a single operation
        sum = fma(temp, temp, sum);
    }

    return sum;
}

int copy_to_gpu_constant(const double *host, size_t count) {
    double * temp;
    hipError_t err = hipGetSymbolAddress((void**)&temp, dev_centers);
    if (err == hipErrorInvalidValue)
        printf("Error: hipErrorInvalidValue\n");
    else if (err == hipErrorInvalidSymbol)
        printf("Error: hipErrorInvalidSymbol\n");
    else if (err == cudaErrorDuplicateVariableName)
        printf("Error: cudaErrorDuplicateVariableName\n");
    if (copy_to_gpu(host, temp, count) != 0) {
        return -1;
    }
    return 0;
}

int copy_from_gpu_constant(double *host, size_t count) {
    double * temp;
    hipError_t err = hipGetSymbolAddress((void**)&temp, dev_centers);
    if (err == hipErrorInvalidValue)
        printf("Error: hipErrorInvalidValue\n");
    else if (err == hipErrorInvalidSymbol)
        printf("Error: hipErrorInvalidSymbol\n");
    else if (err == cudaErrorDuplicateVariableName)
        printf("Error: cudaErrorDuplicateVariableName\n");
    if (copy_from_gpu(host, temp, count) != 0) {
        return -1;
    }
    return 0;
}

int copy_between_gpu_constant(double *host, size_t count) {
    double * temp;
    hipError_t err = hipGetSymbolAddress((void**)&temp, dev_centers);
    if (err == hipErrorInvalidValue)
        printf("Error: hipErrorInvalidValue\n");
    else if (err == hipErrorInvalidSymbol)
        printf("Error: hipErrorInvalidSymbol\n");
    else if (err == cudaErrorDuplicateVariableName)
        printf("Error: cudaErrorDuplicateVariableName\n");
    if (hipMemcpy((void*) host, temp, count, hipMemcpyDeviceToDevice) != hipSuccess)
        return -1;
    return 0;
}

void transpose(double** src, double* dst, int n, int m){
    int i, j;
    for(i=0; i<n; i++){
        for(j=0; j<m; j++){
            dst[j*n + i] = src[i][j];
        }
    }
}

double** create_2D_double_array(int n, int dim) {
    double **arr, *temp;
    temp = (double *)calloc(n * dim, sizeof(double));
    arr = (double **)calloc(n, sizeof(double *));

    for (int i = 0 ; i < n; i++)
        arr[i] = temp + i * dim;

    if (arr == NULL || temp == NULL) {
        fprintf(stderr, "Error in allocation!\n");
        exit(-1);
    }

    return arr;
}

void delete_points(double** ps){
    free(ps);
    ps = NULL;
}

double** init_centers_kpp(double **ps, int n, int k, int dim){
    int i;
    int curr_k = 0;
    int first_i;
    int max, max_i;
    double *distances_from_centers, *temp_distances;
    distances_from_centers = (double*) malloc(sizeof(double)*n);
    double **centers = create_2D_double_array(k,dim);
    temp_distances = (double*) malloc(sizeof(double)*n);
    
    // Initialize with max double
    for (i = 0; i < n; i++)
        distances_from_centers[i] = DBL_MAX;

    srand(time(NULL));

    // Choose a first point
    first_i = rand() % n;
    DPRINTF("First random index: %d", first_i);

    memcpy(centers[curr_k], ps[first_i], dim * sizeof(double));
    DPRINTF("Point 1: (%f, %f)", ps[first_i][0], ps[first_i][1]);
    DPRINTF("Center 1: (%f, %f)", centers[curr_k][0], centers[curr_k][1]);

    while(curr_k < k-1) {
        max = -1;
        max_i = -1;
        for(i=0; i<n; i++){
            DPRINTF("New squared_distance: %f and old min squared_distance: %f", squared_distance(ps[i], centers[curr_k], dim), distances_from_centers[i]);
            temp_distances[i] = MIN(squared_distance(ps[i], centers[curr_k], dim), distances_from_centers[i]);  
            if(temp_distances[i] > max){
                max = temp_distances[i];
                max_i = i;
            }
        }
 
        memcpy(distances_from_centers, temp_distances, n * sizeof(double));
        memcpy(centers[++curr_k], ps[max_i], dim * sizeof(double));
    }
    
    free(temp_distances);
    free(distances_from_centers);
    return centers;
}

__global__
void find_cluster_on_gpu(const double *dev_points,
                         const int n, const int k, const int dim, 
                         double *result_clusters) {

    double min, dist;
    int cluster_it_belongs = 0;
    register int index = get_global_tid();
    register int j;
    const int thread_id = threadIdx.x;
    extern __shared__ double local_points[];
    int block_size = blockDim.x;

    if (index < n){
        for (j = 0; j < dim; ++j){
            local_points[thread_id + j*block_size] = dev_points[index + j];
            // if (index < k)
            //     printf("dev_centers[%d][%d] = %lf\n", index, j, dev_centers[j * k + index]);
        }

        min = DBL_MAX;
        for (j = 0; j < k; ++j){
            result_clusters[j*n + index] = 0.0;
            dist = squared_distance_on_gpu(&local_points[thread_id], &dev_centers[j], block_size, k, dim);
            if (min > dist){
                min = dist;
                cluster_it_belongs = j;
            }
        }
        result_clusters[cluster_it_belongs*n + index] = 1.0;
        for (int j = 0; j < k; j++){
            printf("result_clusters[%d][%d] = %lf --> line[%d]\n", j, index, result_clusters[j*n + index], index+2);
        }
    }
}

__global__
void update_center_on_gpu(const int k, const int dim, 
              double* dev_new_centers,
                          const double* dev_points_in_cluster){
    int j;
    const int index = get_global_tid();

    // do all numbers in k*dim threads 
    if (index < k){
        if (dev_points_in_cluster[index] > 0) {
            #pragma unroll
            for (j = 0; j < dim; j++){
                printf("Before: Dev_new_centers[%d][%d] = %d\n", index, j, dev_new_centers[j*k+index]);
                dev_new_centers[j*k + index] /= dev_points_in_cluster[index];
                printf("After: Dev_new_centers[%d][%d] = %d\n", index, j, dev_new_centers[j*k+index]);
            }
        }
    }
}

__global__
void create_dev_ones(double* dev_ones, int n) {
    int index = get_global_tid();

    if(index < n){
        dev_ones[index] = 1.0;
    }
}

// Just a wrapper function of create_dev_ones to avoid putting that
// function into kmeans_gpu. (create_dev_ones is used in main)
void call_create_dev_ones(double* dev_ones, int n, dim3 gpu_grid, dim3 gpu_block) {
    create_dev_ones<<<gpu_grid,gpu_block>>>(dev_ones, n);
    hipDeviceSynchronize();
}


void swap(double** src, double** dst){
    double *temp = *src;
    *src = *dst;
    *dst = temp;
}

void kmeans_on_gpu(
            double* dev_points,
            int n, int k, int dim,
            double* dev_points_clusters,
            double* dev_points_in_cluster,
            double* dev_new_centers,
            int* dev_check,
            int BLOCK_SIZE, 
            hipblasHandle_t handle,
            hipblasStatus_t stat,
            double* dev_ones,
            double* dev_temp_centers) {

    double alpha = 1.0, beta = 0.0;

    // Calculate grid and block sizes
    int grid_size = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
    dim3 gpu_grid(grid_size, 1);
    dim3 gpu_block(BLOCK_SIZE, 1);
    
    // printf("Grid size : %dx%d\n", gpu_grid.x, gpu_grid.y);
    // printf("Block size: %dx%d\n", gpu_block.x, gpu_block.y);
    // printf("Shared memory size: %ld bytes\n", shmem_size);
    printf("A\n");
    // assign points to clusters - step 1
    find_cluster_on_gpu<<<gpu_grid,gpu_block, BLOCK_SIZE*dim*sizeof(double)>>>(
        dev_points,
        n, k, dim,
        dev_points_clusters);
    hipDeviceSynchronize();
    printf("B\n");
    // update means - step 2
    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                k, dim, n,
                &alpha,
                dev_points_clusters, n,
                dev_points, n,
                &beta,
                dev_new_centers, k);
    // hipDeviceSynchronize();
    printf("C\n");
    hipblasDgemv(handle, HIPBLAS_OP_T,
                n, k,
                &alpha,
                dev_points_clusters, n,
                dev_ones, 1,
                &beta,
                dev_points_in_cluster, 1);
    // hipDeviceSynchronize();
    printf("D\n");
    // Update centers based on counted points
    update_center_on_gpu<<<gpu_grid,gpu_block>>>(
        k, dim,
        dev_new_centers,
        dev_points_in_cluster);
    hipDeviceSynchronize();
    printf("E\n");
    //Check for convergence with CUBLAS
    //dev_new_centers and dev_centers arrays are actually checked for equality
    //No distances are calculated separately for each center point.
    //It seems like its working smoothly so far
    int icheck = 0; //This is used to make it compatible with how the code works now
    double check = 0.0;
    //First subtract the dev_center arrays
    alpha = -1.0;

    hipblasDaxpy(handle, k*dim, &alpha, dev_new_centers, 1, dev_centers, 1);
    // hipDeviceSynchronize();
    //Now find the norm2 of the new_centers
    // hipblasSetPointerMode(handle,HIPBLAS_POINTER_MODE_HOST);
    hipblasDnrm2(handle, k*dim, dev_centers, 1, &check);
    if (!(check > EPS)) icheck = 1;
    copy_to_gpu(&icheck, dev_check, sizeof(int));
    printf("E\n");
    //Update new centers
    // TODO: Swap pointers
    hipDeviceSynchronize();
    if(copy_between_gpu_constant(dev_new_centers, k * dim * sizeof(double)) != 0){
        printf("Error in copy_between_gpu_constant centers\n");
    }
    hipDeviceSynchronize();
    
    printf("F\n");
}