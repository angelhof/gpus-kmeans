#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu_util.h"
#include "hipblas.h" 

#define MIN(X, Y) (((X) < (Y)) ? (X) : (Y))
#define MAX(X, Y) (((X) > (Y)) ? (X) : (Y))

#ifndef EPS
#   define EPS 1.e-6
#endif

// #define DEBUG

#ifdef DEBUG
#define DPRINTF(fmt, args...) \
do { \
    printf("%s, line %u: " fmt "\r\n", __FUNCTION__, __LINE__ , ##args); \
    fflush(stdout); \
} while (0)
#else   
#define DPRINTF(fmt, args...) do{}while(0)
#endif

__device__ int get_global_tid() {
    return (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x*blockDim.y +
        blockDim.x*threadIdx.y + threadIdx.x;
}

double squared_distance(double* ps, double* center, int dim) {
    double sum = 0;

    for (int i = 0; i < dim; i++){
        double temp = center[i] - ps[i];
        sum += temp * temp;
    }

    return sum;
}

__device__
double squared_distance_on_gpu(double* ps, double* center, int dim) {
    double sum = 0;

    for (int i = 0; i < dim; i++){
        double temp = center[i] - ps[i];
        sum += temp * temp;
    }

    return sum;
}

double** create_2D_double_array(int n, int dim) {
    double **arr, *temp;
    temp = (double *)calloc(n * dim, sizeof(double));
    arr = (double **)calloc(n, sizeof(double *));

    for (int i = 0 ; i < n; i++)
        arr[i] = temp + i * dim;

    if (arr == NULL || temp == NULL) {
        fprintf(stderr, "Error in allocation!\n");
        exit(-1);
    }

    return arr;
}

void delete_points(double** ps){
    free(ps);
    ps = NULL;
}

double** init_centers_kpp(double **ps, int n, int k, int dim){
    int i;
    int curr_k = 0;
    int first_i;
    int max, max_i;
    double *distances_from_centers, *temp_distances;
    distances_from_centers = (double*) malloc(sizeof(double)*n);
    double **centers = create_2D_double_array(k,dim);
    temp_distances = (double*) malloc(sizeof(double)*n);
    
    // Initialize with max double
    for (i = 0; i < n; i++)
        distances_from_centers[i] = DBL_MAX;

    srand(time(NULL));

    // Choose a first point
    first_i = rand() % n;
    DPRINTF("First random index: %d", first_i);

    memcpy(centers[curr_k], ps[first_i], dim * sizeof(double));
    DPRINTF("Point 1: (%f, %f)", ps[first_i][0], ps[first_i][1]);
    DPRINTF("Center 1: (%f, %f)", centers[curr_k][0], centers[curr_k][1]);

    while(curr_k < k-1) {
        max = -1;
        max_i = -1;
        for(i=0; i<n; i++){
            DPRINTF("New squared_distance: %f and old min squared_distance: %f", squared_distance(ps[i], centers[curr_k], dim), distances_from_centers[i]);
            temp_distances[i] = MIN(squared_distance(ps[i], centers[curr_k], dim), distances_from_centers[i]);  
            if(temp_distances[i] > max){
                max = temp_distances[i];
                max_i = i;
            }
        }
 
        memcpy(distances_from_centers, temp_distances, n * sizeof(double));
        memcpy(centers[++curr_k], ps[max_i], dim * sizeof(double));
    }
    
    free(temp_distances);
    free(distances_from_centers);
    return centers;
}

__global__
void find_cluster_on_gpu(double *dev_points, double *dev_centers, int n, int k, int dim, 
                         double *result_clusters) {

    double min, dist;
    int cluster_it_belongs;
    int index = get_global_tid();

    int start = index;
    int end = start + 1;

    if (index < n){
        for (int i = start; i < end; i++){
            min = DBL_MAX;
            for (int j = 0; j < k; j++){
                result_clusters[j*n + i] = 0.0;
                dist = squared_distance_on_gpu(&dev_points[i*dim], &dev_centers[j*dim], dim);

                if (min > dist){
                    min = dist;
                    cluster_it_belongs = j;
                }
            }
            // Only 1 in the cluster it belongs and everything else 0
            result_clusters[cluster_it_belongs*n + i] = 1.0;
            // for (int j = 0; j < k; j++){
            //     printf("result_clusters[%d][%d] = %lf --> line[%d]\n", j, i, result_clusters[j*n + i], i+2);
            // }
        }
    }
}

__global__
void update_center_on_gpu(int n, int k, int dim, 
                          double* dev_centers, 
                          double* dev_points_in_cluster,
                          double* dev_temp_centers){
    int i, j;
    int index = get_global_tid();

    int start = index;
    int end = start + 1;

    if (index < k){
        for (i = start; i < end; i++) {
            // printf("dev_points_in_cluster[%d] = %d\n", i, (int)dev_points_in_cluster[i]);
            // for (j = 0; j < dim; j++){
            //     printf("dev_centers[%d][%d] = %lf\n", i, j, dev_centers[j*k + i]);
            // }
            if (dev_points_in_cluster[i] > 0) {
                for (j = 0; j < dim; j++){
                    // FIXME: Two arrays here because of the transposed reslults of CUBLAS
                    dev_temp_centers[i*dim + j] = dev_centers[j*k + i] / (int)dev_points_in_cluster[i];
                }
                // printf("Points in cluster: %d, %d\n", index, dev_points_in_cluster[i]);
            }
            // for (j = 0; j < dim; j++){
            //     printf("new_dev_centers[%d][%d] = %lf\n", i, j, dev_centers[j*k + i]);
            // }
        }
    }
}

__global__
void create_dev_ones(double* dev_ones, int n) {
    int index = get_global_tid();

    if(index < n){
        dev_ones[index] = 1.0;
    }
}

// Just a wrapper function of create_dev_ones to avoid putting that
// function into kmeans_gpu. (create_dev_ones is used in main)
void call_create_dev_ones(double* dev_ones, int n, dim3 gpu_grid, dim3 gpu_block) {
    create_dev_ones<<<gpu_grid,gpu_block>>>(dev_ones, n);
    hipDeviceSynchronize();
}

void kmeans_on_gpu(
            double* dev_points,
            double* dev_centers,
            int n, int k, int dim,
            double* dev_points_clusters,
            double* dev_points_in_cluster,
            double* dev_new_centers,
            int* dev_check,
            int BLOCK_SIZE, 
            //CUBLAS Shit
            hipblasHandle_t handle,
            hipblasStatus_t stat,
            double* dev_ones,
            double* dev_temp_centers) {

    double alpha = 1.0, beta = 0.0;

    // Calculate grid and block sizes
    int grid_size = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
    dim3 gpu_grid(grid_size, 1);
    dim3 gpu_block(BLOCK_SIZE, 1);
    
    // printf("Grid size : %dx%d\n", gpu_grid.x, gpu_grid.y);
    // printf("Block size: %dx%d\n", gpu_block.x, gpu_block.y);
    // printf("Shared memory size: %ld bytes\n", shmem_size);

    // assign points to clusters - step 1
    find_cluster_on_gpu<<<gpu_grid,gpu_block>>>(
        dev_points,
        dev_centers,
        n, k, dim,
        dev_points_clusters);
    hipDeviceSynchronize();
    
    // update means - step 2
    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
                k, dim, n,
                &alpha,
                dev_points_clusters, n,
                dev_points, dim,
                &beta,
                dev_new_centers, k);
    hipDeviceSynchronize();

    hipblasDgemv(handle, HIPBLAS_OP_T,
                n, k,
                &alpha,
                dev_points_clusters, n,
                dev_ones, 1,
                &beta,
                dev_points_in_cluster, 1);
    hipDeviceSynchronize();

    // Update centers based on counted points
    update_center_on_gpu<<<gpu_grid,gpu_block>>>(
        n, k, dim,
        dev_new_centers,
        dev_points_in_cluster,
        dev_temp_centers);
    hipDeviceSynchronize();
    
    //Check for convergence with CUBLAS
    //dev_new_centers and dev_centers arrays are actually checked for equality
    //No distances are calculated separately for each center point.
    //It seems like its working smoothly so far
    int icheck = 0; //This is used to make it compatible with how the code works now
    double check = 0.0;
    //First subtract the dev_center arrays
    alpha = -1.0;
    hipblasDaxpy(handle, k*dim, &alpha, dev_temp_centers, 1, dev_centers, 1);
    hipDeviceSynchronize();
    //Now find the norm2 of the new_centers
    // hipblasSetPointerMode(handle,HIPBLAS_POINTER_MODE_HOST);
    hipblasDnrm2(handle, k*dim, dev_centers, 1, &check);
    hipDeviceSynchronize();
    if (!(check > EPS)) icheck = 1;
    copy_to_gpu(&icheck, dev_check, sizeof(int));
    
    //Update new centers
    hipMemcpy(dev_centers, dev_temp_centers, sizeof(double)*k*dim, hipMemcpyDeviceToDevice);
}