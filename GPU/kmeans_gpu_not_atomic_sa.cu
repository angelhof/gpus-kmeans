#include "hip/hip_runtime.h"
#include <string.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "gpu_util.h"
#include "kmeans_util_sa.h"
#include "kmeans_minibatch_gpu.h" 
#include "kmeans_sa_gpu.h" 
#include "hipblas.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/* gpu parameters */

//#define GRID_SIZE 16
//#define BLOCK_SIZE 256

#define DIMENSION 2
#define KMEANS1
#define SA
#define MINI_BATCHES1

int main(int argc, char *argv[]) {
    
    int n, k, i, j;
    int dim = 2;
    double **points;
    
    int BLOCK_SIZE = 256; //Default
    if (argc > 1) BLOCK_SIZE = atoi(argv[1]);
    
    //The second input argument should be the dataset filename
    FILE *in;
    if (argc > 2) {
        in = fopen(argv[2], "r");
    } else {
        in = stdin;
    }
    
    //Parse file
    fscanf(in, "%d %d %d\n", &n ,&k, &dim);
    points = create_2D_double_array(n, dim);
    for (i =0; i<n; i++) {
        for (j=0; j<dim; j++) {
            fscanf(in, "%lf", &points[i][j]);
        }
    }
    fclose(in);
        
    printf("Input Read successfully \n");
    
    //Create CUBLAS Handles
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    
    // Calculate grid and block sizes
    int grid_size = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
    dim3 gpu_grid(grid_size, 1);
    dim3 gpu_block(BLOCK_SIZE, 1);
    int thread_num = grid_size * BLOCK_SIZE;
    
    printf("Grid size : %dx%d\n", gpu_grid.x, gpu_grid.y);
    printf("Block size: %dx%d\n", gpu_block.x, gpu_block.y);
    
    clock_t start = clock();
    
    double **centers;
    printf("Initializing Centers...\n");
    centers = init_centers_kpp(points, n, k, dim);
    printf("Initializing Centers done\n");
    
    // start algorithm
    double *points_clusters;

    points_clusters = (double *)calloc(n*k, sizeof(double));
    
    // GPU allocations
    double *dev_centers, *dev_points, *dev_centers_of_points;
    double *dev_points_help;
    double *dev_new_centers;
    double *dev_points_clusters;
    int *dev_points_clusters_old;
    double *dev_points_in_cluster;
    double *dev_ones;
    //RNG CUDA States
    hiprandState* devStates;

    dev_centers = (double *) gpu_alloc(k*dim*sizeof(double));
    dev_points = (double *) gpu_alloc(n*dim*sizeof(double));
    dev_centers_of_points = (double *) gpu_alloc(n*dim*sizeof(double));
    dev_points_in_cluster = (double *) gpu_alloc(k*sizeof(double));
    dev_points_clusters = (double *) gpu_alloc(n*k*sizeof(double));
    dev_points_clusters_old = (int *) gpu_alloc(n*sizeof(int)); //Used for SA SAKM
    dev_new_centers = (double *) gpu_alloc(k*dim*sizeof(double));
    dev_ones = (double *) gpu_alloc(n*sizeof(double));
    dev_points_help = (double *) gpu_alloc(n*sizeof(double));
    
    printf("GPU allocs done \n");
    //Nullify the counter array
    hipMemset(dev_points_in_cluster, 0x0, k*sizeof(double));
    call_create_dev_ones(dev_ones, n, gpu_grid, gpu_block);
    // Transpose points and centers for cublas
    // TODO: Transpose at cublas in gpu
    double * staging_points = (double*) calloc(n*dim, sizeof(double));
    double * staging_centers = (double*) calloc(k*dim, sizeof(double));
    transpose(points, staging_points, n, dim);
    transpose(centers, staging_centers, k, dim);

    // Copy points to GPU
    if (copy_to_gpu(staging_points, dev_points, n*dim*sizeof(double)) != 0) {
        printf("Error in copy_to_gpu points\n");
        return -1;
    }

    // Copy centers to GPU
    if (copy_to_gpu(staging_centers, dev_centers, k*dim*sizeof(double)) != 0) {
        printf("Error in copy_to_gpu centers\n");
        return -1;
    }

    //Setup Random States
    hipMalloc(&devStates,  thread_num * sizeof(hiprandState));
    setup_RNG_states(devStates, gpu_grid, gpu_block);

    //Init the result_cluster arrays once 
    init_point_clusters(dev_points, dev_centers, 
                        n, k, dim, 
                        gpu_grid, gpu_block,
                        dev_points_clusters, dev_points_clusters_old, 
                        devStates);

    // FIXME: For now we pass TWO matrices for centers, one normal and 
    //        one transposed. The transposed can be omitted by doing some
    //        changes in Step 1 of K-Means.
    double *dev_temp_centers,  *dev_temp_points_clusters;
    dev_temp_centers = (double *) gpu_alloc(k*dim*sizeof(double));
    dev_temp_points_clusters = (double *) gpu_alloc(n*k*sizeof(double));

    int step = 1;
    int check = 0;
    int* dev_check = (int *) gpu_alloc(sizeof(int));
    double* dev_cost = (double *) gpu_alloc(sizeof(double));

    printf("Centers Init \n");
    //Debug
    for(i=0;i<k;i++){
        for(j=0;j<k*dim;j+=k)
            printf("%lf,\t", staging_centers[j + i]);
        printf("\n");
    }
    srand(unsigned(time(NULL)));

    /*
            SA & K-MEANS ALGORITHM
        
    */
#ifdef SA
    //SA config
    //SA starting temperature should be set so that the probablities of making moves on the very
    //first iteration should be very close to 1.
    //Start temp of 100 seems to be working good for the tested datasets
    double start_temp = 100.0;
    double temp = start_temp;
    int eq_iterations = 160;
    double best_cost = DBL_MAX;


    //SA loop
    printf("Starting SA on GPU \n");
    int eq_counter = 0;
    while(eq_counter < eq_iterations) {
        
        //printf("SA Temp: %lf \n", temp);
        //Sample solution space with SA
        double cost = kmeans_on_gpu_SA(
                    dev_points,
                    dev_centers,
                    n, k, dim,
                    dev_points_clusters,
                    dev_points_clusters_old,
                    dev_points_in_cluster,
                    dev_centers_of_points,
                    dev_new_centers,
                    dev_check,
                    gpu_grid, 
                    gpu_block, 
                    handle,
                    stat,
                    dev_ones,
                    dev_points_help, 
                    dev_temp_centers, 
                    devStates, 
                    temp);

        //Acceptance checks
        if (cost <= best_cost){
            //Accept the solution immediately        
            //Found better solution
            best_cost = cost;
            //printf("Found Better Solution: %lf Temp %lf\n", cost, temp);
            hipMemcpy(dev_centers, dev_new_centers, sizeof(double)*k*dim, hipMemcpyDeviceToDevice);
            //Storing global best to temp_centers
            hipMemcpy(dev_temp_centers, dev_new_centers, sizeof(double)*k*dim, hipMemcpyDeviceToDevice);
            hipMemcpy(dev_temp_points_clusters, dev_points_clusters, sizeof(double)*k*n, hipMemcpyDeviceToDevice);
            
        } else {
            //Accept the solution with probability
            double accept_factor = 1.0; // The larger the factor the less the probability becomes
            //Increasing the factor is equivalent with decreasing the start_temp

            double prob = exp(-accept_factor*(cost - best_cost)/start_temp);
            double uniform = rand() / (RAND_MAX + 1.);
            if (prob > uniform){
                //Accept solution as the current one
                // printf("Accepting with Prob: %lf Diff %lf\n", prob, cost - best_cost);
                hipMemcpy(dev_centers, dev_new_centers, sizeof(double)*k*dim, hipMemcpyDeviceToDevice);
            }
        }
        step += 1;
        eq_counter++;
    }
    //Storing global best to temp_centers
    hipMemcpy(dev_new_centers, dev_temp_centers, sizeof(double)*k*dim, hipMemcpyDeviceToDevice);
    hipMemcpy(dev_points_clusters, dev_temp_points_clusters, sizeof(double)*k*n, hipMemcpyDeviceToDevice);
    printf("SA Steps %d \n", step);
#endif

    /*
        DEFAULT K-MEANS ALGORITHM
    
    */
#ifdef KMEANS
    
    step = 0;
    printf("Proper KMeans Algorithm \n");
    while (!check) {
        double cost = kmeans_on_gpu(
                        dev_points,
                        dev_centers,
                        n, k, dim,
                        dev_points_clusters,
                        dev_points_in_cluster,
                        dev_centers_of_points, 
                        dev_new_centers,
                        dev_check,
                        BLOCK_SIZE,
                        handle,
                        stat,
                        dev_ones,
                        dev_points_help, 
                        dev_temp_centers);
        
        copy_from_gpu(&check, dev_check, sizeof(int));
        //printf("Step %4d Check: %d Cost: %lf \n", step, check, cost);
        step += 1;
    }
    printf("KMeans algorithm steps %d \n", step);
#endif

#ifdef MINI_BATCHES

    int BATCH_SIZE = 64;
        
    //Create batch arrays
    double *batch_points, *batch_points_clusters;
    int *batch_points_clusters_old;
    batch_points = (double *) gpu_alloc(BATCH_SIZE*dim*sizeof(double));
    batch_points_clusters = (double *) gpu_alloc(BATCH_SIZE*k*sizeof(double));
    batch_points_clusters_old = (int *) gpu_alloc(BATCH_SIZE*sizeof(int));

    step = 0;
    double cost = DBL_MAX;
    printf("Mini Batch KMeans Algorithm \n");
    while (step < 50000) {
        // Serial minibatch loops internally no need for further repeats here
        // cost = kmeans_serial_MINIBATCH(
        //                 dev_points,
        //                 dev_centers,
        //                 dev_new_centers,
        //                 dev_points_in_cluster, 
        //                 n, k, dim,
        //                 dev_points_clusters,
        //                 devStates, 
        //                 handle);
        //break;

        cost = kmeans_on_gpu_MINIBATCH(
                          dev_points,
                          dev_centers,
                          n, k, dim,
                          dev_points_clusters,
                          dev_points_clusters_old,
                          dev_points_in_cluster,
                          dev_new_centers,
                          dev_check,
                          //CUBLAS Shit
                          handle,
                          stat,
                          dev_ones,
                          dev_points_help,
                          dev_temp_centers,
                          devStates, 
                          //BATCH arrays
                          BATCH_SIZE, 
                          batch_points, 
                          batch_points_clusters, 
                          batch_points_clusters_old);
        

        //printf("Step %4d Cost: %lf \n", step, cost);
        step += 1;
    }

    hipFree(batch_points);
    hipFree(batch_points_clusters);
    hipFree(batch_points_clusters_old);

    printf("Mini Batch KMeans algorithm steps %d \n", step);
#endif


    //Post Processing
    double eval = evaluate_solution(dev_points, dev_new_centers, dev_points_clusters, 
                  dev_centers_of_points, dev_points_help, 
                  n, k, dim, 
                  gpu_grid, gpu_block, 
                  handle, stat);

    printf("Final Solution Value: %lf \n", eval);

    double time_elapsed = (double)(clock() - start) / CLOCKS_PER_SEC;
    printf("Total Time Elapsed: %lf seconds\n", time_elapsed);
    
    FILE *f;
    //Store Performance metrics
    //For now just the time elapsed, in the future maybe we'll need memory GPU memory bandwidth etc...
    f = fopen("log.out", "w");
    fprintf(f, "Time Elapsed: %lf ", time_elapsed);
    fclose(f);
    
    // print & save results
    copy_from_gpu(staging_centers, dev_new_centers, k*dim*sizeof(double));
    f = fopen("centers.out", "w");
    printf("Centers:\n");
    for (i = 0; i < k; i++) {
        for (j = 0; j < dim; j++){
            printf("%lf ", staging_centers[j*k + i]);
            fprintf(f, "%lf ", staging_centers[j*k + i]);
        }
        printf("\n");
        fprintf(f, "\n");
    }
    fclose(f);
    
    //Store Mapping Data in case we need it
    copy_from_gpu(points_clusters, dev_points_clusters, n*k*sizeof(double));
    f = fopen("point_cluster_map.out", "w");
    for (i =0;i<k;i++){
        for (j=0;j<n;j++){
            fprintf(f, "%lf ", points_clusters[i*n + j]);    
        }
        fprintf(f, "\n");
    }
    
    fclose(f);
    
    // GPU clean
    gpu_free(dev_centers);
    gpu_free(dev_new_centers);
    gpu_free(dev_temp_centers);
    gpu_free(dev_points);
    gpu_free(dev_points_clusters);
    gpu_free(dev_temp_points_clusters);
    gpu_free(dev_points_in_cluster);
    gpu_free(dev_centers_of_points);
    gpu_free(devStates);

    stat = hipblasDestroy(handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS destruction failed\n");
        return EXIT_FAILURE;
    }

    // clear and exit
    delete_points(points);
    delete_points(centers);
    free(points_clusters);
    return 0;
}
